#include "hip/hip_runtime.h"
#include<slicesample.h>

	__global__ 
void slicesample(double *u,double *y,double *theta,long int N,int order,int num_samples,double *theta_0,double elim)
{
	const int tid = blockDim.x*blockIdx.x+threadIdx.x;
	const int tt = blockDim.x*gridDim.x;	
		
	if(tid<tt)
	{
		unsigned int seed_normal = hash(tid);
		unsigned int seed_uniform = hash(tid);
		thrust::default_random_engine rng_normal(seed_normal);
		thrust::default_random_engine rng_uniform(seed_uniform);

		thrust::random::experimental::normal_distribution<double> dist_norm(0.0, 1.0);
		thrust::random::uniform_real_distribution<double> dist_uniform(0.0, 1.0);
		
		//Populate first theta value with initial theta_0 vector
		for(int jj=0;jj<2*(order+1);jj++)
		{		
			theta[N*jj+tid] = theta_0[jj];	
			theta[N*jj+(tid+tt)] = theta_0[jj];
			//printf("theta[%ld] = %f\n",N*jj+tid,theta[N*jj+tid]);		


		}		

		//Start proposal distribution with standard deviation sigma
		double width = 0.01;
		double y_test[100];
		double theta_new[4],theta_new_l[4],theta_new_r[4],theta_prime[4];
		
		int ii = tid+tt;
		for(int jj=0;jj<(2*(order+1));jj++)
		{
			theta_new[jj] = theta[N*jj+ii];
			//printf("theta_new[%d] = %f\n",jj,theta_new[jj]);
		}

		filter(theta_new,y_test,u,num_samples,order,N);
		double pstar = p_ratio(num_samples,elim,y,y_test);
				
		while(ii<N)
		{
			
			double Puprime = pstar+std::log(dist_uniform(rng_uniform));
		
			int kk=1;
			while(kk < 2*(order+1))
			{
				for(int jj=0;jj<2*(order+1);jj++)
				{
					theta_new[jj] = theta[N*jj+ii];
					theta_new_l[jj] = theta[N*jj+ii];
					theta_new_r[jj] = theta[N*jj+ii];
					theta_prime[jj] = theta[N*jj+ii];
				}
				
				double bit = dist_uniform(rng_uniform);
				theta_new_l[kk] = theta_new[kk]-bit*width;
				theta_new_r[kk] = theta_new[kk]+bit*width;
				
				//Step out to span target density
				filter(theta_new_l,y_test,u,num_samples,order,N);
				while(p_ratio(num_samples,elim,y,y_test)>Puprime)
				{
					theta_new_l[kk] = theta_new_l[kk]-width;
					filter(theta_new_l,y_test,u,num_samples,order,N);
				}

				filter(theta_new_r,y_test,u,num_samples,order,N);
				while(p_ratio(num_samples,elim,y,y_test)>Puprime)
				{
					theta_new_r[kk] = theta_new_r[kk]+width;
					filter(theta_new_l,y_test,u,num_samples,order,N);
				}

				int stepcount = 0;

				while(1)
				{
					stepcount++;
					
					thrust::random::uniform_real_distribution<double> dist_uniform_2(theta_new_l[kk],theta_new_r[kk]);
					theta_prime[kk] = dist_uniform_2(rng_uniform);
					filter(theta_prime,y_test,u,num_samples,order,N);
					pstar = p_ratio(num_samples,elim,y,y_test);

					if(pstar>Puprime)
					{
						//printf("BREAK\n");
						break;
					}
						
					else
					{
						if(theta_prime[kk]>theta_new[kk])
							theta_new_r[kk] = theta_prime[kk];
						else if(theta_prime[kk]<theta_new[kk])
							theta_new_l[kk] = theta_prime[kk];
						else
							printf("ERROR\n");
					}
				}
				theta[N*kk+ii] = theta_prime[kk];
				kk=kk+1;
				

			}

			
			ii = ii+tt;
			if(ii>=N)
				break;
			for(int jj=0;jj<2*(order+1);jj++)
				theta[N*jj+ii] = theta[N*jj+(ii-tt)];
		}
	}

			

}

