#include<mcmc.h>

void mcmc(double *u,double *y,double *theta,long int N,int order, int num_samples,double *theta_0,double elim)
{

	double *d_u,*d_y,*d_y_test,*d_theta,*d_theta_0;

	int u_size = num_samples*sizeof(double);
	long int theta_size = N*2*(order+1)*sizeof(double);
	int theta_0_size = 2*(order+1)*sizeof(double);

	printf("N = %ld\n",N);
	printf("order = %d\n",order);
	printf("num_samples = %d\n",num_samples);
	printf("theta_size = %ld\n",theta_size);

	hipMalloc((void**)&d_u, u_size ); 
	hipMalloc((void**)&d_y, u_size ); 
	hipMalloc((void**)&d_theta, theta_size );
	hipMalloc((void**)&d_y_test, u_size );
	hipMalloc((void**)&d_theta_0,theta_0_size);


	hipMemcpy(d_u, u, u_size, hipMemcpyHostToDevice );
	hipMemcpy(d_y, y, u_size, hipMemcpyHostToDevice );
	hipMemcpy(d_theta_0,theta_0,theta_0_size,hipMemcpyHostToDevice);

	slicesample<<<8,128>>>(d_u,d_y,d_theta,N,order,num_samples,d_theta_0,elim);

	hipMemcpy( theta, d_theta, theta_size, hipMemcpyDeviceToHost ); 


	hipFree(d_u);
	hipFree(d_y);
	hipFree(d_theta);
	hipFree(d_theta_0);
	hipFree(d_y_test);

	hipDeviceReset();

	return ;
}
