#include "hip/hip_runtime.h"
#include<metropolis.h>

	__global__ 
void metropolis(double *u,double *y,double *theta,long int N,int order,int num_samples,double *theta_0,double elim)
{
	const int tid = blockDim.x*blockIdx.x+threadIdx.x;
	const int tt = blockDim.x*gridDim.x;	
	double y_test[50];
	if(tid<tt)
	{
		unsigned int seed_normal = hash(tid);
		unsigned int seed_uniform = hash(tid);
		thrust::default_random_engine rng_normal(seed_normal);
		thrust::default_random_engine rng_uniform(seed_uniform);

		thrust::random::experimental::normal_distribution<double> dist_norm(0, 1);
		thrust::random::uniform_real_distribution<double> dist_uniform(0, 1);
		
		for(int ii=0;ii<2*(order+1);ii++)		
			theta[N*ii+tid] = theta_0[ii];		

		double sigma = 1.0;

		int kk=0;
		int win_width = 100;
		int win_accept = 0;
		int accepted=0;
		int flg=0;
		int ii=tid+tt;
				
		while(ii<N)
		{
			//Generate Proposal
		
			for(int jj=0;jj<2*(order+1);jj++)
			{
				theta[N*jj+ii] = theta[N*jj+ii-tt] + sigma*dist_norm(rng_normal);
			}
			theta[ii] = 1.0;			

			filter(&theta[ii],y_test,u,num_samples,order,N);
			//Calculate P_ratio
			//the ratio of the probability of the new theta guess to the old theta guess
			//P(theta_new)/P(theta_old)
			
			double Pr = p_ratio(num_samples,elim,y,y_test);


			if(Pr <= dist_uniform(rng_uniform))
			{
				//Don't accept the new theta value
				for(int jj=0;jj<2*(order+1);jj++)
					theta[N*jj+ii] = theta[N*jj+ii-tt];
			}
			else
			{	
				//Accept new theta value
				accepted++;				
				win_accept++;
			}
			kk++;
			//Check burnin
			if(kk%win_width ==0&&flg==0)
			{
				if((double)win_accept/win_width>0.25)
				{
					sigma=sigma*1.2;
				}
				else if((double)win_accept/win_width<0.2)
				{
					sigma = sigma/1.2;
				}
				if(kk%10000==0)
				{
					flg=1;
					printf("acceptance rate from thread %d = %f\n",tid,(double)win_accept/win_width);
					win_width = win_width*2;

				}
				for(int jj=0;jj<2*(order+1);jj++)
					theta[N*jj+tid] = theta[N*jj+ii-tt];
				win_accept=0;
				ii=tid;
				accepted=0;
				
			}
			
			ii +=tt;		
		}
		printf("acceptance rate from thread %d = %f\n",tid,(float)accepted/(N/(tt)));
		printf("sigma from thread %d = %f\n",tid,sigma);			


	}

}

