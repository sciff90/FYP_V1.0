#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <fstream>

#define Nthreads 256
#define PI 3.141592

	__device__
void filter_out(double *theta,double *y,const double *u,int num_samples,int order)
{
	int ii,jj;
	int N = 20000000;
	double *a = &theta[0];
	double *b = &theta[N*(order+1)];

	for(ii=0;ii<num_samples;ii++)y[ii] = 0.0;	
	
	
	for (ii = 0; ii < (num_samples-order); ii++)
	{
		for (jj = 1; jj <= order; jj++)
		{
			if(ii-jj>=0)
				y[ii] += - a[jj*N]*y[ii-jj];
			else
				y[ii] += 0;
		}
		for (jj = 0; jj <= order; jj++)
		{
			if(ii-jj>=0)
				y[ii] += b[jj*N]*u[ii-jj];
			else
				y[ii] +=0;
		}

		y[ii] = y[ii]/a[0];		
	}
}
	__device__
unsigned int hash(unsigned int a)
{
	a = (a+0x7ed55d16) + (a<<12);
	a = (a^0xc761c23c) ^ (a>>19);
	a = (a+0x165667b1) + (a<<5);
	a = (a+0xd3a2646c) ^ (a<<9);
	a = (a+0xfd7046c5) + (a<<3);
	a = (a^0xb55a4f09) ^ (a>>16);
	return a;
}

	__global__ 
void mcmc_kernel(double *u,double *y,double *theta,int N,int order,int num_samples,double *theta_0,double elim)
{
	const int tid = blockDim.x*blockIdx.x+threadIdx.x;
	const int tt = blockDim.x*gridDim.x;	
	double y_test[50];
	if(tid<tt)
	{
		unsigned int seed_normal = hash(tid);
		thrust::default_random_engine rng_normal(seed_normal);
		//thrust::default_random_engine rng_uniform(seed_uniform);

		thrust::random::experimental::normal_distribution<double> dist_norm(0, 1);
		//thrust::random::experimental::uniform_distribution<double> dist_uniform(0, 1);
		
		for(int ii=0;ii<2*(order+1);ii++)		
			theta[N*ii+tid] = theta_0[ii];		

		double sigma = 0.01;

		int kk=0;
		int accepted=0;
		int flg=0;
		for(int ii=tid+tt;ii<N;ii +=tt)
		{
			//Generate Proposal
			
			for(int jj=0;jj<2*(order+1);jj++)
			{
				theta[N*jj+ii] = theta[N*jj+ii-tt] + sigma*dist_norm(rng_normal);
			}
			theta[ii] = 1.0;			

			filter_out(&theta[ii],y_test,u,num_samples,order);
			float max_diff = 0;
			int max_loc = 0;
			float diff;			
			
			for(int jj=0;jj<num_samples-order;jj++)
			{
				diff = abs(y[jj]-y_test[jj]);
				if(diff > max_diff) 
				{
					max_diff = diff;
					max_loc = jj;
				}
			}
			if(max_diff>elim)
			{
				//printf("flagged max_diff = %f at postion %d\n",max_diff,max_loc);
				//printf("y[max_loc] = %f y_test[max_loc] = %f\n",y[max_loc],y_test[max_loc]);
				//printf("a1=  %f\n",theta[N*1+ii]);
				for(int jj=0;jj<2*(order+1);jj++)
					theta[N*jj+ii] = theta[N*jj+ii-tt];
			}
			else
			{
				accepted++;				
			}
			kk++;
			if(kk%100==0 && kk!=0 && flg==0)
			{
				if((double)accepted/100>0.4)
				{
					sigma=sigma*1.2;					
				}
				else if((double)accepted/100<0.3)
				{
					sigma = sigma/1.2;
				}
				else
				{
					flg=1;				
					printf("sigma = %f\n",sigma);
				}
				//printf("a_rate = %f\n",(double)accepted/1000);
				kk=0;
				accepted=0;
				ii = tid+tt;
				
			}
			

						
		}

	}

}

void mcmc(double *u,double *y,double *theta,int N,int order, int num_samples,double *theta_0,double elim)
{

	double *d_u,*d_y,*d_y_test,*d_theta,*d_theta_0;

	int u_size = num_samples*sizeof(double);
	int theta_size = N*2*(order+1)*sizeof(double);
	int theta_0_size = 2*(order+1)*sizeof(double);

	printf("N = %d\n",N);
	printf("order = %d\n",order);
	printf("num_samples = %d\n",num_samples);
	printf("theta_size = %d\n",theta_size);

	hipMalloc((void**)&d_u, u_size ); 
	hipMalloc((void**)&d_y, u_size ); 
	hipMalloc((void**)&d_theta, theta_size );
	hipMalloc((void**)&d_y_test, u_size );
	hipMalloc((void**)&d_theta_0,theta_0_size);


	hipMemcpy(d_u, u, u_size, hipMemcpyHostToDevice );
	hipMemcpy(d_y, y, u_size, hipMemcpyHostToDevice );
	hipMemcpy(d_theta_0,theta_0,theta_0_size,hipMemcpyHostToDevice);

	mcmc_kernel<<<8,128>>>(d_u,d_y,d_theta,N,order,num_samples,d_theta_0,elim);

	hipMemcpy( theta, d_theta, theta_size, hipMemcpyDeviceToHost ); 


	hipFree(d_u);
	hipFree(d_y);
	hipFree(d_theta);
	hipFree(d_theta_0);
	hipFree(d_y_test);

	return ;
}
